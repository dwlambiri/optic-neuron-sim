
#include <hip/hip_runtime.h>
__global__  void propagation_algo_cuda(
        int N,
        float* tox_prod,
        short* axons,
        unsigned char* blue,
        float* cMap2,
        const float* cMap1,
        float* detox,
        const float* centers,
        float dInside,
        float dOutside,
        int lowerLimit,
        int upperLimit,
        float deathThreashold,
        float amountReleasedOnDeath,
        float  outsideDetox,
        bool   algo,
        float* deathThr)
{
	//int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int yidx = blockIdx.y * blockDim.y + threadIdx.y;

    int index = yidx*N+ xidx;
    int indexUp = (yidx-1)*N + xidx;
    int indexDown = (yidx+1)*N + xidx;
    int indexLeft = yidx*N + xidx -1;
    int indexRight = yidx*N + xidx+1;

    const short aliveAxon_c = 1;
    const short deadAxon_c  = 2;
    const short noAxon_c    = -1;

    const int stopProduction_c = 0;
    const unsigned char blueSky_c = 255;

    
    if((indexUp < lowerLimit) || (indexDown < lowerLimit) || (indexLeft < lowerLimit) || (indexRight < lowerLimit)) {
        return;
    }

    if((indexUp > upperLimit) || (indexDown > upperLimit) || (indexLeft > upperLimit) || (indexRight > upperLimit)) {
        return;
    }
   
    int centerIndex = centers[index];

    float extraAmount = 0;

    if(axons[index] == aliveAxon_c){
      if(cMap1[index] > deathThr[index]) {
        axons[index] = deadAxon_c;
        extraAmount = amountReleasedOnDeath;
        tox_prod[index] = stopProduction_c;
        detox[index] = outsideDetox;
        blue[index] = blueSky_c;
      }
    }
    
    if(centerIndex > 0 && axons[centerIndex-1] == deadAxon_c && tox_prod[index] > 0) {
        extraAmount = amountReleasedOnDeath;
        tox_prod[index] = stopProduction_c;
        detox[index] = outsideDetox;
        blue[index] = blueSky_c;
    }

    float di = dOutside;

    if(centerIndex > 0 && axons[centerIndex-1] == aliveAxon_c) {
       di = dInside; 
    }

    float t = cMap1[index];

    if(algo == true) {
        cMap2[index] = t +

                    (cMap1[indexUp] - t) * ((centers[indexUp]== -1)?0:di) +
                    (cMap1[indexDown] - t) * ((centers[indexDown]== -1)?0:di) +
                    (cMap1[indexLeft] - t) * ((centers[indexLeft]== -1)?0:di) +
                    (cMap1[indexRight] - t) * ((centers[indexRight]== -1)?0:di) +
                    tox_prod[index] + extraAmount;
    }
    else {
        cMap2[index] = t +
                (cMap1[indexUp] - t) * (di) +
                (cMap1[indexDown] - t) * (di) +
                (cMap1[indexLeft] - t) * (di) +
                (cMap1[indexRight] - t) * (di) +
                 tox_prod[index] + extraAmount;  
    }

    cMap2[index] *= detox[index];
               
/*
    if(cMap2[index] > deathThreashold && tox_prod[index] > 0) {
        cMap2[index] = amountReleasedOnDeath;
        tox_prod[index] = 0; 
        return;
    }
*/      
    
}
