
#include <hip/hip_runtime.h>
__global__  void propagation_algo_cuda(
        int N,
        float* tox_prod,
        short* axons,
        float* blue,
        float* cMap2,
        const float* cMap1,
        float* detox,
        const float* centers,
        float dInside,
        float dOutside,
        int lowerLimit,
        int upperLimit,
        float deathThreashold,
        float amountReleasedOnDeath,
        float  outsideDetox)
{
	//int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int yidx = blockIdx.y * blockDim.y + threadIdx.y;

    int index = yidx*N+ xidx;
    int indexUp = (yidx-1)*N + xidx;
    int indexDown = (yidx+1)*N + xidx;
    int indexLeft = yidx*N + xidx -1;
    int indexRight = yidx*N + xidx+1;

    
    if((indexUp < lowerLimit) || (indexDown < lowerLimit) || (indexLeft < lowerLimit) || (indexRight < lowerLimit)) {
        return;
    }

    if((indexUp > upperLimit) || (indexDown > upperLimit) || (indexLeft > upperLimit) || (indexRight > upperLimit)) {
        return;
    }
   
    int centerIndex = centers[index];

    if(axons[index] == 1){
      if(cMap2[index] > deathThreashold) {
        cMap2[index] = amountReleasedOnDeath;
        tox_prod[index] = 0; 
        axons[index] = 2;
        blue[index] = 64;
        detox[index] = outsideDetox;
        return;
      }
    }

    if(centerIndex > 0 && axons[centerIndex-1] ==2 && tox_prod[index] > 0) {
        cMap2[index] = amountReleasedOnDeath;
        tox_prod[index] = 0;
        detox[index] = outsideDetox;
        blue[index] = 64;
        return;
    }

    float di = dOutside;

    if(centerIndex > 0 && axons[centerIndex-1] == 1) {
       di = dInside; 
    }
    float t = cMap1[index];
    cMap2[index] = t +
                (cMap1[indexUp] - t) * di +
                (cMap1[indexDown] - t) * di +
                (cMap1[indexLeft] - t) * di +
                (cMap1[indexRight] - t) * di +
                tox_prod[index];

    cMap2[index] *= detox[index];
/*
    if(cMap2[index] > deathThreashold && tox_prod[index] > 0) {
        cMap2[index] = amountReleasedOnDeath;
        tox_prod[index] = 0; 
        return;
    }
*/      
    
}
