
#include <hip/hip_runtime.h>
__global__  void propagation_algo_cuda(
        int N,
        float* tox_prod,
        short* axons,
        float* blue,
        float* cMap2,
        const float* cMap1,
        const float* detox,
        const float* centers,
        double dInside,
        double dOutside,
        int lowerLimit,
        int upperLimit)
{
	//int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int yidx = blockIdx.y * blockDim.y + threadIdx.y;

    int index = yidx*N+ xidx;
    int indexUp = (yidx-1)*N + xidx;
    int indexDown = (yidx+1)*N + xidx;
    int indexLeft = yidx*N + xidx -1;
    int indexRight = yidx*N + xidx+1;

    
    if((indexUp < lowerLimit) || (indexDown < lowerLimit) || (indexLeft < lowerLimit) || (indexRight < lowerLimit)) {
        return;
    }

    if((indexUp > upperLimit) || (indexDown > upperLimit) || (indexLeft > upperLimit) || (indexRight > upperLimit)) {
        return;
    }
   
    if(centers[index] < 0) {
         return;
    }

    float di = dOutside;

    if(centers[index] > 0 ) {
       di = dInside; 
    }

    float t = cMap1[index];
    cMap2[index] = t +
                (cMap1[indexUp] - t) * di +
                (cMap1[indexDown] - t) * di +
                (cMap1[indexLeft] - t) * di +
                (cMap1[indexRight] - t) * di +
                tox_prod[index];

    cMap2[index] *= detox[index];
/*
    if(cMap2[index] > 22 && tox_prod[index] > 0) {
        cMap2[index] = 10000;
        tox_prod[index] = 0; 
        return;
    }
*/
       
    if(axons[index] == 1){
      if(cMap2[index] > 22) {
        cMap2[index] = 10000;
        tox_prod[index] = 0; 
        axons[index] = 2;
        blue[index] = 1;
      }
      return;
    }
    
    int centerIndex = centers[index];

    if(centerIndex <= 0){
        return;
    }
       
    if(tox_prod[index] > 0 && axons[centerIndex-1] == 2 && cMap2[index] > 22) {
        cMap2[index] = 10000;
        tox_prod[index] = 0;
        blue[index] = 1;
    }
}
