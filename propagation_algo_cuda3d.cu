
#include <hip/hip_runtime.h>
__global__  void propagation_algo_cuda3d(
        int N,
        float* tox_prod,
        short* axons,
        unsigned char* blue,
        float* cMapResult,
        const float* cMapUp,
        const float* cMap,
        const float* cMapDown,
        float* detox,
        const float* centers,
        float dInside,
        float dOutside,
        int lowerLimit,
        int upperLimit,
        float deathThreashold,
        float amountReleasedOnDeath,
        float  outsideDetox,
        bool   algo,
        float* deathThr,
        bool top,
        bool bottom,
        bool injury)
{
	//int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int yidx = blockIdx.y * blockDim.y + threadIdx.y;

    int index = yidx*N+ xidx;
    int indexUp = (yidx-1)*N + xidx;
    int indexDown = (yidx+1)*N + xidx;
    int indexLeft = yidx*N + xidx -1;
    int indexRight = yidx*N + xidx+1;

    const short aliveAxon_c = 1;
    const short deadAxon_c  = 2;
    const short noAxon_c    = -1;

    const int stopProduction_c = 0;
    const unsigned char blueSky_c = 255;

    
    if((indexUp < lowerLimit) || (indexDown < lowerLimit) || (indexLeft < lowerLimit) || (indexRight < lowerLimit)) {
        return;
    }

    if((indexUp > upperLimit) || (indexDown > upperLimit) || (indexLeft > upperLimit) || (indexRight > upperLimit)) {
        return;
    }
   
    int centerIndex = centers[index];

    float extraAmount = 0;

    if(axons[index] == aliveAxon_c){
      if(top == true && cMap[index] > deathThr[index]) {
        axons[index] = deadAxon_c;
        extraAmount = amountReleasedOnDeath;
        tox_prod[index] = stopProduction_c;
        detox[index] = outsideDetox;
        blue[index] = blueSky_c;
      }
    }
    
    if(centerIndex > 0 && axons[centerIndex-1] == deadAxon_c && tox_prod[index] > 0) {
        extraAmount = amountReleasedOnDeath;
        tox_prod[index] = stopProduction_c;
        detox[index] = outsideDetox;
        blue[index] = blueSky_c;
    }

    float di = dOutside;

    if(centerIndex > 0 && axons[centerIndex-1] == aliveAxon_c) {
       di = dInside; 
    }

    float t = cMap[index];

    if(algo == true) {
        cMapResult[index] = t +
                    (cMap[indexUp] - t) * ((centers[indexUp]== -1)?0:di) +
                    (cMap[indexDown] - t) * ((centers[indexDown]== -1)?0:di) +
                    (cMap[indexLeft] - t) * ((centers[indexLeft]== -1)?0:di) +
                    (cMap[indexRight] - t) * ((centers[indexRight]== -1)?0:di);

                    if((top == false) && (bottom == false)) {
                        cMapResult[index] += (cMapUp[index] - t)*di +
                                             (cMapDown[index] -t)*di;
                    }
                    if((top == true) && (bottom == false)) {
                        cMapResult[index] += (cMapDown[index] -t)*di;
                    }
                    if((top == false) && (bottom == true)) {
                        cMapResult[index] += (cMapUp[index] -t)*di;
                    }
                    if(injury == true) {
                        cMapResult[index] += tox_prod[index] + extraAmount;
                    }
    }
    else {
        cMapResult[index] = t +
                (cMap[indexUp] - t) * (di) +
                (cMap[indexDown] - t) * (di) +
                (cMap[indexLeft] - t) * (di) +
                (cMap[indexRight] - t) * (di); 
        if((top == false) && (bottom == false)) {
            cMapResult[index] += (cMapUp[index] - t)*di +
                    (cMapDown[index] -t)*di;
        }
        if((top == true) && (bottom == false)) {
            cMapResult[index] += (cMapDown[index] -t)*di;
        }
        if((top == false) && (bottom == true)) {
            cMapResult[index] += (cMapUp[index] -t)*di;
        }
        if(injury == true) {
            cMapResult[index] += tox_prod[index] + extraAmount;
        }
    }

    cMapResult[index] *= detox[index];
               
/*
    if(cMapResult[index] > deathThreashold && tox_prod[index] > 0) {
        cMapResult[index] = amountReleasedOnDeath;
        tox_prod[index] = 0; 
        return;
    }
*/      
    
}
